#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void matadd(int *l,int *m, int *n)
{
    int x=blockIdx.x; //By using blockIdx.x, we can determine the position of the current block along the x-axis of the grid.
    int y=blockIdx.y;
    int id=gridDim.x * y +x; // Since the grid can have multiple dimensions (in this case, it's a 2D grid), we need to calculate a unique identifier for each thread that takes into account both the x and y coordinates
    n[id]=l[id]+m[id];
}
int main()
{
    int a[2][3];
    int b[2][3];
    int c[2][3];
    int *d,*e,*f;
    int i,j;
    
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
            {
                a[i][j]=i+j;
            }
    }
    
        for(i=0;i<2;i++)
        {
            for(j=0;j<3;j++)
                {
                    b[i][j]=i+j;
                }
        }
     // (void **)&d is used to pass the address of the pointer d to cudaMalloc. 
     // sizeof(int) gives the size of an integer in bytes. Multiplying it by 2 * 3 gives the total number of bytes required to store the matrix.
 
    hipMalloc((void **)&d,2*3*sizeof(int));  // responsible for allocating device memory on the GPU.
    hipMalloc((void **)&e,2*3*sizeof(int));
    hipMalloc((void **)&f,2*3*sizeof(int));
 
    // When you allocate device memory using cudaMalloc, you are requesting a block of memory on the GPU with a specified size. 
    //This memory is used to store data that will be processed by the GPU kernels.
    
 hipMemcpy(d,a,2*3*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,2*3*sizeof(int),hipMemcpyHostToDevice);
    
dim3 grid(3,2);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */

    matadd<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,2*3*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nSum of two matrices:\n ");
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);
    return 0;
}
