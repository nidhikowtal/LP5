#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void matproduct(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k;
  
n[2*y+x]=0;
for(k=0;k<2;k++)
   {
    n[2*y+x]=n[2*y+x]+l[2*y+k]*m[2*k+x];
   }
}

int main()
{
    int row1=2, col1=2, row2=2, col2=2;
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

    //printf("\n Enter elements of first matrix of size 2*3\n");
    for(i=0;i<2;i++)
    {
        for(j=0;j<2;j++)
            {
                a[i][j]=i+j;
            }
    }
 for(i=0;i<2;i++)
    {
        for(j=0;j<2;j++)
        {
              printf("%d\t",a[i][j]);
        }
        printf("\n");
    }
    //printf("\n Enter elements of second matrix of size 3*2\n");
        for(i=0;i<2;i++)
        {
            for(j=0;j<2;j++)
                {
                    b[i][j]=i+j;
                }
        }
 for(i=0;i<2;i++)
    {
        for(j=0;j<2;j++)
        {
              printf("%d\t",b[i][j]);
        }
        printf("\n");
    }

    hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */

    matproduct<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nProduct of two matrices:\n ");
    for(i=0;i<2;i++)
    {
        for(j=0;j<2;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}